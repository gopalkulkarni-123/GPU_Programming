#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <math.h>
#include <iomanip>
#include <fstream>
#include <string>
#include <hip/hip_runtime.h>
#include <vector>

// Global variables for grid dimensions and cut coordinates
std::string FILE_NAME;
const double EPS = 1e-3;

// Structure to represent a block of the grid
struct BlockOfGrid {
    int xMin;
    int xMax;
    int yMin;
    int yMax;
    float alpha = 0.75;
    float dx = 1.0;
    float dy = 1.0;
    float dt = 0.1;
    float r_x = alpha * dt / (2 * dx * dx);
    float r_y = alpha * dt / (2 * dy * dy);
    float tempDiff = 100.0;
    float maxTempDiff = 100.0;
    float localGrid[50][50];

    // Constructor with built-in boundary trimming
    BlockOfGrid(int x_min, int x_max, int y_min, int y_max)
        : xMin(x_min),
          xMax(x_max),
          yMin(y_min),
          yMax(y_max) {
        for (int i = 0; i < xMax - xMin; ++i) {
            for (int j = 0; j < yMax - yMin; ++j) {
                localGrid[i][j] = 0;
            }
        }
    }

    __device__ void computeNextStateAll(const float grid[50][50]) {
        maxTempDiff = 0.0;
        for (int i = std::max(1, xMin); i < std::min(xMax, 100 - 1); ++i) {
            for (int j = std::max(1, yMin); j < std::min(yMax, 100 - 1); ++j) {
                tempDiff = r_x * (grid[i + 1][j] - 2 * grid[i][j] + grid[i - 1][j]) + r_y * (grid[i][j + 1] - 2 * grid[i][j] + grid[i][j - 1]);
                localGrid[i - xMin][j - yMin] = grid[i][j] + tempDiff;
                maxTempDiff = std::max(maxTempDiff, tempDiff);
            }
        }
        if (xMin > 0 && xMax < 100 && yMin > 0 && yMax < 100) {
            return;
        }

        if (xMin == 0) {
            for (int j = yMin; j < yMax; ++j) {
                localGrid[0][j - yMin] = 100;
            }
        }
        if (xMax == 100) {
            for (int j = yMin; j < yMax; ++j) {
                localGrid[xMax - xMin - 1][j - yMin] = 100;
            }
        }
        if (yMin == 0) {
            for (int i = xMin; i < xMax; ++i) {
                localGrid[i - xMin][0] = 0;
            }
        }
        if (yMax == 100) {
            for (int i = xMin; i < xMax; ++i) {
                localGrid[i - xMin][yMax - yMin - 1] = 0;
            }
        }
    }

    __device__ void updateGlobalGrid(float grid[25][25]) {
        for (int i = xMin; i < xMax; ++i) {
            for (int j = yMin; j < yMax; ++j) {
                grid[i][j] = localGrid[i - xMin][j - yMin];
            }
        }
    }
};

inline void checkCudaError(hipError_t result, const char* file, int line) {
    if (result != hipSuccess) {
        std::cerr << "CUDA error at " << file << ":" << line << " - " << hipGetErrorString(result) << std::endl;
        exit(1);
    }
}

#define CUDA_CHECK(result) checkCudaError(result, __FILE__, __LINE__)

void initializeGrid(float grid){
    // Initialize the grid with some values
    for (int i = 0; i < 100; ++i) {
        for (int j = 0; j < 100; ++j) {
            grid[i][j] = 0.0f;
        }
    }
    // Set boundary conditions
    for (int i = 0; i < 100; ++i) {
        grid[i][0] = 100.0f;  // Left boundary
        grid[i][100 - 1] = 100.0f;  // Right boundary
    }
    for (int j = 0; j < 100; ++j) {
        grid[0][j] = 100.0f;  // Top boundary
        grid[100 - 1][j] = 100.0f;  // Bottom boundary
    }
}

__global__ void simulateHeat(BlockOfGrid* individualBlock, float grid[50][50]){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = idx; i < numBlocks; i += blockDim.x * gridDim.x) {
        individualBlock[i].computeNextStateAll(grid);  // Pass the grid here
    }
}

int main() {
    float mainGrid [100][100];
    initializeGrid(mainGrid);

    BlockOfGrid vectorOfBlocks[4] = {
        {0, 50, 0, 50},
        {0, 50, 50, 100},
        {50, 100, 0, 50},
        {50, 100, 50, 100}
    };

    BlockOfGrid* deviceBlocks;
    hipMalloc(&deviceBlocks, sizeof(vectorOfBlocks) * 4);
    //hipMalloc(&mainGrid, sizeof(float) * 100 * 100);

    //Copy data to device
    hipMemcpy(deviceBlocks, vectorOfBlocks, sizeof(MatrixBlock) * 4, hipMemcpyDeviceToHost);

    processBlocks<<<1, 4>>>simulateHeat(deviceBlocks, )



