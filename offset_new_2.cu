#include <iostream>
#include <hip/hip_runtime.h>

__global__ void offsetDevice(float* testArray, int offsetValue, int numOfElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x + offsetValue;
    testArray[i] = testArray[i] * 2;
}

inline hipError_t checkCuda(hipError_t result) {
#if defined(DEBUG) || defined(_DEBUG)
    if (result != cudaSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
        assert(result == cudaSuccess);
    }
#endif
    return result;
}

int main() {
    int nMB = 4;
    int n = nMB * 1024 * 1024 / sizeof(float);
    int blockSize = 256;

    hipEvent_t startEvent, stopEvent;
    float ms;

    //int* hostArray = new int[n];

    //for (int i = 0; i < n; ++i) {
    //    hostArray[i] = i;
    //}

    // Allocate, copy, execute, and free the device memory
    float* deviceArray;
    int size = n * 33 * sizeof(float);
    int numBlocks = (n + blockSize - 1) / blockSize;

    checkCuda(hipMalloc(&deviceArray, size));

    checkCuda(hipEventCreate(&startEvent));
    checkCuda(hipEventCreate(&stopEvent));

    offsetDevice<<<numBlocks, blockSize>>>(deviceArray, 0, n);

    for (int i = 0; i < 34; ++i) {
        checkCuda(hipMemset(deviceArray, 0, n * sizeof(float))); // Corrected memset size

        checkCuda(hipEventRecord(startEvent, 0));
        offsetDevice<<<numBlocks, blockSize>>>(deviceArray, i, n);
        checkCuda(hipEventRecord(stopEvent, 0));
        checkCuda(hipEventSynchronize(stopEvent));

        checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));

        std::cout << (2 * n * sizeof(int) / (1024.0f * 1024.0f * 1024.0f)) / (ms / 1000.0f) << std::endl;
    }

    checkCuda( hipEventDestroy(startEvent) );
    checkCuda( hipEventDestroy(stopEvent) );
    checkCuda(hipFree(deviceArray));
    //delete[] hostArray;

    return 0;
}
