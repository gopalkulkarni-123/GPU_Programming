#include <iostream>
#include <hip/hip_runtime.h>

__global__ void offsetDevice(float* testArray, int offsetValue, int numOfElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x + offsetValue;
    testArray[i] = testArray[i] * 2;
}

void display(float* array, int n){
    for (int i = 0; i < n; ++i){
        std::cout << "i : " << array[i] << std::endl;
    }
}

inline hipError_t checkCuda(hipError_t result) {
#if defined(DEBUG) || defined(_DEBUG)
    if (result != cudaSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
        assert(result == cudaSuccess);
    }
#endif
    return result;
}

int main() {
    int nMB = 4;
    int n = nMB * 1024 * 1024 / sizeof(float);
    int blockSize = 256;

    hipEvent_t startEvent, stopEvent;
    float ms;

    float* hostArray = new float[n];

    for (int i = 0; i < n; ++i) {
        hostArray[i] = i;
    }

    // Allocate, copy, execute, and free the device memory
    float* deviceArray;
    int size = n * 33 * sizeof(float);
    int numBlocks = (n + blockSize - 1) / blockSize;

    checkCuda(hipMalloc(&deviceArray, size));

    checkCuda(hipEventCreate(&startEvent));
    checkCuda(hipEventCreate(&stopEvent));

    checkCuda(hipMemcpy(deviceArray, hostArray, size, hipMemcpyHostToDevice));

    offsetDevice<<<numBlocks, blockSize>>>(deviceArray, 0, n);

    for (int i = 0; i < 34; ++i) {
        checkCuda(hipMemset(deviceArray, 0, n * sizeof(float))); // Corrected memset size

        checkCuda(hipEventRecord(startEvent, 0));
        offsetDevice<<<numBlocks, blockSize>>>(deviceArray, i, n);
        checkCuda(hipEventRecord(stopEvent, 0));
        checkCuda(hipEventSynchronize(stopEvent));

        checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));

        checkCuda(hipMemcpy(hostArray, deviceArray, size, hipMemcpyDeviceToHost));

        std::cout << (2 * n * sizeof(int) / (1024.0f * 1024.0f * 1024.0f)) / (ms / 1000.0f) << std::endl;
    }

    checkCuda( hipEventDestroy(startEvent) );
    checkCuda( hipEventDestroy(stopEvent) );
    checkCuda(hipFree(deviceArray));
    //display(hostArray, n);
    delete[] hostArray;

    return 0;
}
