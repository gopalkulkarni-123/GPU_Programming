#include <iostream>
#include <hip/hip_runtime.h>

__global__ void strideDevice(float* testArray, int strideValue, int numOfElements) {
    int i = (blockDim.x * blockIdx.x + threadIdx.x) * strideValue ;
    testArray[i] = testArray[i] * 2;
}

inline
hipError_t checkCuda(hipError_t result){
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %sn", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

int main (){

    int nMB = 4;
    int n = nMB * 1024 * 1024/sizeof(float);
    std::cout << "n value is " << n << std::endl;
    //int offsetValue = 1;
    int blockSize = 256;

    hipEvent_t startEvent, stopEvent;
    float ms;

    //int* hostArray = new int[n];

    //for (int i = 0; i < n; ++i){
    //    hostArray[i] = i;
    //}

    //Allocate, copy, execute, and free the device memory
    float* deviceArray;
    int size = n * 66 * sizeof(float);
    int numBlocks = (n + blockSize -1)/blockSize;

    checkCuda(hipMalloc(&deviceArray, size));

    checkCuda(hipEventCreate(&startEvent));
    checkCuda(hipEventCreate(&stopEvent));

    strideDevice<<<numBlocks, blockSize>>>(deviceArray, 0, n);

    for (int i = 1; i < 34; ++i){
      checkCuda (hipMemset(deviceArray, 0, n * sizeof(float)));

      checkCuda(hipEventRecord(startEvent, 0));
      strideDevice<<<numBlocks, blockSize>>>(deviceArray, i, n);
      checkCuda(hipEventRecord(stopEvent, 0));
      checkCuda(hipEventSynchronize(stopEvent));
      
      checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
      std::cout << (2 * n * sizeof(int) / (1024.0f * 1024.0f * 1024.0f)) / (ms / 1000.0f) << std::endl;
    }
    checkCuda( hipEventDestroy(startEvent) );
    checkCuda( hipEventDestroy(stopEvent) );
    checkCuda(hipFree(deviceArray));

    //delete[] hostArray;
    return 0;
}