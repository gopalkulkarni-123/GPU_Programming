#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

#define N 100  // Global grid size
#define NUM_BLOCKS 4  // Number of blocks
#define BLOCK_SIZE 50  // Size of each block (50x50)

struct BlockOfGrid {
    int xMin, xMax, yMin, yMax, width;
    float* localGrid;  // Points to a subregion in the global grid

    __host__ __device__
    BlockOfGrid(int x_min = 0, int x_max = 0, int y_min = 0, int y_max = 0, int gridWidth = 0, float* gridPtr = nullptr)
        : xMin(x_min), xMax(x_max), yMin(y_min), yMax(y_max), width(gridWidth), localGrid(gridPtr) {}

    // Utility functions
    __device__ float& at(int i, int j) {
        return localGrid[i * width + j];
    }

    __device__ void compute() {
        for (int i = 0; i < (xMax - xMin); ++i) {
            for (int j = 0; j < (yMax - yMin); ++j) {
                at(i, j) = at(i, j) + 2.0f;
                //localGrid[i * width + j] = localGrid[i * width + j] + 2.0f;
            }
        }
    }
};

__global__ void processBlocks(BlockOfGrid* blocks, int numBlocks) {
    int idx = threadIdx.x;
    if (idx < numBlocks) {
        blocks[idx].compute();
    }
}

int main() {
    float* mainGrid = new float[N * N];
    float* hostLocalGrids = new float[NUM_BLOCKS * BLOCK_SIZE * BLOCK_SIZE];

    // Initialize mainGrid
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            if (i == 0 || j == 0 || i == N - 1 || j == N - 1) {
                mainGrid[(i * N) + j] = 100.0f;
            } else {
                mainGrid[(i * N) + j] = 0.0f;
            }
        }
    }
    mainGrid[(50 * 100 + 50)] = 205.0f;
    // Displays the main grid
    /*for (int i = 0; i < 100 * 100; ++i){
        std::cout << "mainGrid ["<< i <<"] " << mainGrid[i] << std::endl;
    }
    std::cout << "--------------------------------------------" << std::endl;*/

    // Define block metadata and copy data from mainGrid to hostLocalGrids
    BlockOfGrid hostBlocks[NUM_BLOCKS];
    for (int b = 0; b < NUM_BLOCKS; ++b) {
        int xMin = (b / 2) * BLOCK_SIZE;
        int xMax = xMin + BLOCK_SIZE;
        int yMin = (b % 2) * BLOCK_SIZE;
        int yMax = yMin + BLOCK_SIZE;
        int width = yMax - yMin;
        float* localGridPtr = &hostLocalGrids[b * BLOCK_SIZE * BLOCK_SIZE];

        // Copy corresponding block from mainGrid
        for (int i = 0; i < BLOCK_SIZE; ++i) {
            for (int j = 0; j < BLOCK_SIZE; ++j) {
                //int globalI = xMin + i;
                //int globalJ = yMin + j;
                localGridPtr[i * BLOCK_SIZE + j] = mainGrid[(b * BLOCK_SIZE*BLOCK_SIZE) + (i * BLOCK_SIZE + j)];
                //std::cout << "[" << i * BLOCK_SIZE + j << "]" << "[" << globalI * N + globalJ << "]" << std::endl;
                //std::cout <<"b = " << b <<"; i = " << i << "; j = " << j <<  "; i * BLOCK_SIZE + j = " << i * BLOCK_SIZE + j
                // <<  "; (b+1) * i * BLOCK_SIZE + j = " << (b*BLOCK_SIZE*BLOCK_SIZE) + (i * BLOCK_SIZE + j) << std::endl;  
            }
        }

        hostBlocks[b] = BlockOfGrid(xMin, xMax, yMin, yMax, BLOCK_SIZE, localGridPtr);
    }

    //comparison of localGrid and mainGrid
    /*std::cout << "Befor computation" << std::endl;
    for (int i = 0; i < 100 * 100; ++i){
        std::cout << "Grid ["<< i <<"] " << hostLocalGrids[i] << "  MainGrid [" << i <<"] " << mainGrid[i] << std::endl;
    }*/

    /*for (int i = 0; i < NUM_BLOCKS; ++i){
        std::cout << "xMin: " << hostBlocks[i].xMin << ", yMin: " << hostBlocks[i].yMin
         <<  ", xMax: " << hostBlocks[i].xMax <<  ", yMax: " << hostBlocks[i].yMax << std::endl;
    }*/

    // Allocate memory on device
    float* deviceLocalGrids;
    hipMalloc(&deviceLocalGrids, sizeof(float) * NUM_BLOCKS * BLOCK_SIZE * BLOCK_SIZE);
    hipMemcpy(deviceLocalGrids, hostLocalGrids, sizeof(float) * NUM_BLOCKS * BLOCK_SIZE * BLOCK_SIZE, hipMemcpyHostToDevice);

    BlockOfGrid* deviceBlocks;
    for (int b = 0; b < NUM_BLOCKS; ++b) {
        hostBlocks[b].localGrid = deviceLocalGrids + b * BLOCK_SIZE * BLOCK_SIZE;
    }
    hipMalloc(&deviceBlocks, sizeof(BlockOfGrid) * NUM_BLOCKS);
    hipMemcpy(deviceBlocks, hostBlocks, sizeof(BlockOfGrid) * NUM_BLOCKS, hipMemcpyHostToDevice);

    // Launch kernel
    processBlocks<<<1, NUM_BLOCKS>>>(deviceBlocks, NUM_BLOCKS);
    hipDeviceSynchronize();

    // Copy back result
    hipMemcpy(hostLocalGrids, deviceLocalGrids, sizeof(float) * NUM_BLOCKS * BLOCK_SIZE * BLOCK_SIZE, hipMemcpyDeviceToHost);

    float* d_input;
    float* d_output;
    hipMalloc(&d_input, sizeof(float) * NUM_BLOCKS * BLOCK_SIZE * BLOCK_SIZE);
    hipMemcpy(d_input, hostLocalGrids, sizeof(float) * NUM_BLOCKS * BLOCK_SIZE * BLOCK_SIZE, hipMemcpyHostToDevice);
    hipMalloc(&d_output, sizeof(float));

    // 2. Temporary storage
    void* d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;

    // Get temp storage size
    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_input, d_output,
                        NUM_BLOCKS * BLOCK_SIZE * BLOCK_SIZE);
    hipMalloc(&d_temp_storage, temp_storage_bytes);

    // 3. Run max reduction
    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_input, d_output,
                        NUM_BLOCKS * BLOCK_SIZE * BLOCK_SIZE);

    // 4. Copy result to host
    float maxValue;
    hipMemcpy(&maxValue, d_output, sizeof(float), hipMemcpyDeviceToHost);
    std::cout << "\n Max grid value across all blocks: " << maxValue << "\n";

    // Cleanup CUB allocations
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_temp_storage);

    // Copy back results to mainGrid for visualization
    for (int b = 0; b < NUM_BLOCKS; ++b) {
        //int xMin = hostBlocks[b].xMin;
        //int yMin = hostBlocks[b].yMin;
        float* localGridPtr = &hostLocalGrids[b * BLOCK_SIZE * BLOCK_SIZE];

        for (int i = 0; i < BLOCK_SIZE; ++i) {
            for (int j = 0; j < BLOCK_SIZE; ++j) {
                //int globalI = xMin + i;
                //int globalJ = yMin + j;
                mainGrid[(b * BLOCK_SIZE*BLOCK_SIZE) + (i * BLOCK_SIZE + j)] = localGridPtr[i * BLOCK_SIZE + j];
            }
        }
    }

    // Print sample 
    /*for (int i = 0; i < NUM_BLOCKS; ++i) {
        std::cout << "Block " << i << " sample (10,10): "
                  << hostLocalGrids[i * BLOCK_SIZE * BLOCK_SIZE + 10 * BLOCK_SIZE + 10] << "\n";
    }*/

    //After computing
    for (int i = 0; i < 100 * 100; ++i){
        std::cout << "Grid ["<< i <<"] " << hostLocalGrids[i] << std::endl;
    }

    // Cleanup
    hipFree(deviceLocalGrids);
    hipFree(deviceBlocks);
    delete[] hostLocalGrids;
    delete[] mainGrid;

    return 0;
}
